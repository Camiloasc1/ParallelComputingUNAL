
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void PartialMM(double *A, double *B, double *C, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    double c = 0.0;
#pragma unroll
    for (unsigned int k = 0u; k < N; ++k) {
        c += A[i * N + k] * B[k * N + j];
    }
    C[i * N + j] = c;
}

int main(int argc, char *argv[]) {

    unsigned int N = 1024u;
    if (argc > 1) {
        N = (unsigned int) atoi(argv[1]);
    }
    if (N < 32)
        N = 32;
    srand((unsigned int) time(NULL));

    unsigned int size = N * N * sizeof(double);
    double *h_A, *h_B, *h_C;
    double *d_A, *d_B, *d_C;

    h_A = (double *) malloc(size);
    h_B = (double *) malloc(size);
    h_C = (double *) malloc(size);

    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);

    for (unsigned int k = 0u; k < N * N; ++k) {
        h_A[k] = rand();
        h_B[k] = rand();
        h_C[k] = 0.0;
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    dim3 threads(32, 32); // 32*32=1024
    dim3 grid(N / threads.x, N / threads.y);

    PartialMM<<<grid, threads>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Size\t\t%dx%d\n", N, N);
//    printf("Result:");
//    for (unsigned int k = 0u; k < N * N; ++k) {
//        if (k % N == 0)
//            printf("\n");
//        printf("%f ", h_C[k]);
//    }

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    return EXIT_SUCCESS;
}
