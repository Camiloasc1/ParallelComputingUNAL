#include "hip/hip_runtime.h"
#define CHUNK 1 //Run CHUNK blocks, each with 1024 threads (or with the specified argument) and check error
#define LOG 1024 // Print progress each LOG iterations
#define LIMIT 1024*1024 // LIMIT of iterations

#include "../common.h"

__device__ double atomicAdd(double *, double);

__global__ void LeibnizPI(double *sum, unsigned long offset) {

    __shared__ double partialSum;

    if (threadIdx.x == 0) {
        partialSum = 0.0;
    }

    __syncthreads();

    unsigned long n = (offset * CHUNK + blockIdx.x) * blockDim.x + threadIdx.x;
    double val = ((n % 2 == 0) ? 1.0 : -1.0) / ((n << 1) + 1);
    atomicAdd(&partialSum, val);

    __syncthreads();

    if (threadIdx.x == 0) {
        *sum += partialSum;
    }
}

__device__ double atomicAdd(double *address, double val) {
    unsigned long long int *address_as_ull =
            (unsigned long long int *) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

int main(int argc, char *argv[]) {
    unsigned int digits;
    unsigned int threads;
    double precision;
    getParams(argc, argv, &threads, &digits, &precision);

    double h_sum = 0.0;
    double *d_sum;

    hipMalloc((void **) &d_sum, sizeof(double));

    hipMemcpy(d_sum, &h_sum, sizeof(double), hipMemcpyHostToDevice);

    double pi = 0.0, error = 1.0;
    unsigned long i = 0;
    while (error > precision && i < LIMIT) {
        //@formatter:off
        LeibnizPI<<<CHUNK, threads>>>(d_sum, i);
        //@formatter:on
        hipDeviceSynchronize();

        hipMemcpy(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);

        pi = 4.0 * h_sum;
        error = getError(pi);
        printLog(precision, pi, error, ++i);
    }

    hipFree(d_sum);

    hipDeviceReset();

    return EXIT_SUCCESS;
}
